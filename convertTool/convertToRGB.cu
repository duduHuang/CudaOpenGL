#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "convertToRGB.h"

__global__ static void convertToRGBKernel(const uint16_t *pV210, uint16_t *tt, int nSrcWidth,
    int nDstWidth, int nDstHeight) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
    uint4 pF;
    if (tid < (nSrcWidth / 8) && tidd < nDstHeight) {
        int j = tidd * nSrcWidth;
        int k = tid * 8;
        pF.x = (uint32_t)pV210[j + k + 0] + ((uint32_t)pV210[j + k + 1] << 16);
        pF.y = (uint32_t)pV210[j + k + 2] + ((uint32_t)pV210[j + k + 3] << 16);
        pF.z = (uint32_t)pV210[j + k + 4] + ((uint32_t)pV210[j + k + 5] << 16);
        pF.w = (uint32_t)pV210[j + k + 6] + ((uint32_t)pV210[j + k + 7] << 16);

        v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
        y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10) * 1164;
        u0 = (uint32_t)(pF.x & 0x000003FF);
        y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20) * 1164;
        u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
        y1 = (uint32_t)(pF.y & 0x000003FF) * 1164;
        u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
        y3 = (uint32_t)((pF.z & 0x000FFC00) >> 10) * 1164;
        v1 = (uint32_t)(pF.z & 0x000003FF);
        y5 = (uint32_t)((pF.w & 0x3FF00000) >> 20) * 1164;
        v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
        y4 = (uint32_t)(pF.w & 0x000003FF) * 1164;

        k = tid * 18;
        j *= 9;
        j /= 4;
        int r = 1596 * v0 - 891648, g = 813 * v0 + 392 * u0 - 542464, b = 2017 * u0 - 1107200;
        tt[j + k + 0] = (y0 + r) / 1000;
        tt[j + k + 1] = (y0 - g) / 1000;
        tt[j + k + 2] = (y0 + b) / 1000;

        tt[j + k + 3] = (y1 + r) / 1000;
        tt[j + k + 4] = (y1 - g) / 1000;
        tt[j + k + 5] = (y1 + b) / 1000;

        r = 1596 * v1 - 891648, g = 813 * v1 + 392 * u1 - 542464, b = 2017 * u1 - 1107200;
        tt[j + k + 6] = (y2 + r) / 1000;
        tt[j + k + 7] = (y2 - g) / 1000;
        tt[j + k + 8] = (y2 + b) / 1000;

        tt[j + k + 9] = (y3 + r) / 1000;
        tt[j + k + 10] = (y3 - g) / 1000;
        tt[j + k + 11] = (y3 + b) / 1000;

        r = 1596 * v2 - 891648, g = 813 * v2 + 392 * u2 - 542464, b = 2017 * u2 - 1107200;
        tt[j + k + 12] = (y4 + r) / 1000;
        tt[j + k + 13] = (y4 - g) / 1000;
        tt[j + k + 14] = (y4 + b) / 1000;

        tt[j + k + 15] = (y5 + r) / 1000;
        tt[j + k + 16] = (y5 - g) / 1000;
        tt[j + k + 17] = (y5 + b) / 1000;
    }
}

void convertToRGB(uint16_t *dpSrc, uint16_t *dpDst, int nSrcWidth, int nDstWidth, int nDstHeight, hipStream_t stream) {
    dim3 blocks(32, 16, 1);
    dim3 grids((nSrcWidth + blocks.x - 1) / blocks.x, (nDstHeight + blocks.y - 1) / blocks.y, 1);
    convertToRGBKernel << <grids, blocks, 0, stream >> > (dpSrc, dpDst, nSrcWidth, nDstWidth, nDstHeight);
}

__global__ static void convertToRGBTestKernel(const uint16_t *pV210, uint8_t *tt, int nSrcWidth,
    int nDstWidth, int nDstHeight) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
    uint4 pF;
    uint16_t tt1[6];
    
    if (tid < (nSrcWidth / 8) && tidd < nDstHeight) {
        int j = tidd * nSrcWidth;
        int k = tid * 8;
        pF.x = (uint32_t)pV210[j + k + 0] + ((uint32_t)pV210[j + k + 1] << 16);
        pF.y = (uint32_t)pV210[j + k + 2] + ((uint32_t)pV210[j + k + 3] << 16);
        pF.z = (uint32_t)pV210[j + k + 4] + ((uint32_t)pV210[j + k + 5] << 16);
        pF.w = (uint32_t)pV210[j + k + 6] + ((uint32_t)pV210[j + k + 7] << 16);

        v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
        y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10) * 1164;
        u0 = (uint32_t)(pF.x & 0x000003FF);
        y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20) * 1164;
        u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
        y1 = (uint32_t)(pF.y & 0x000003FF) * 1164;
        u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
        y3 = (uint32_t)((pF.z & 0x000FFC00) >> 10) * 1164;
        v1 = (uint32_t)(pF.z & 0x000003FF);
        y5 = (uint32_t)((pF.w & 0x3FF00000) >> 20) * 1164;
        v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
        y4 = (uint32_t)(pF.w & 0x000003FF) * 1164;

        k = tid * 18;
        j *= 9;
        j /= 4;
        int r = 1596 * v0 - 891648, g = 813 * v0 + 392 * u0 - 542464, b = 2017 * u0 - 1107200;
        tt[j + k + 0] = (y0 + r) * 0.249 / 1000;
        tt[j + k + 1] = (y0 - g) * 0.249 / 1000;
        tt[j + k + 2] = (y0 + b) * 0.249 / 1000;

        tt[j + k + 3] = (y1 + r) * 0.249 / 1000;
        tt[j + k + 4] = (y1 - g) * 0.249 / 1000;
        tt[j + k + 5] = (y1 + b) * 0.249 / 1000;

        r = 1596 * v1 - 891648, g = 813 * v1 + 392 * u1 - 542464, b = 2017 * u1 - 1107200;
        tt[j + k + 6] = (y2 + r) * 0.249 / 1000;
        tt[j + k + 7] = (y2 - g) * 0.249 / 1000;
        tt[j + k + 8] = (y2 + b) * 0.249 / 1000;

        tt[j + k + 9] = (y3 + r) * 0.249 / 1000;
        tt[j + k + 10] = (y3 - g) * 0.249 / 1000;
        tt[j + k + 11] = (y3 + b) * 0.249 / 1000;

        r = 1596 * v2 - 891648, g = 813 * v2 + 392 * u2 - 542464, b = 2017 * u2 - 1107200;
        tt[j + k + 12] = (y4 + r) * 0.249 / 1000;
        tt[j + k + 13] = (y4 - g) * 0.249 / 1000;
        tt[j + k + 14] = (y4 + b) * 0.249 / 1000;

        tt[j + k + 15] = (y5 + r) * 0.249 / 1000;
        tt[j + k + 16] = (y5 - g) * 0.249 / 1000;
        tt[j + k + 17] = (y5 + b) * 0.249 / 1000;
    }
}

void convertToRGBTest(uint16_t *dpSrc, uint8_t *dpDst, int nSrcWidth, int nDstWidth, int nDstHeight,
    hipStream_t stream) {
    dim3 blocks(32, 16, 1);
    dim3 grids((nSrcWidth + blocks.x - 1) / blocks.x, (nDstHeight + blocks.y - 1) / blocks.y, 1);
    convertToRGBTestKernel << <grids, blocks, 0, stream >> > (dpSrc, dpDst, nSrcWidth, nDstWidth, nDstHeight);
}

__global__ static void convertVToRGBKernel(const uint16_t *pV210, uint8_t *tt1, int nSrcWidth,
    int nDstWidth, int nDstHeight, int *lookupTable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
    uint16_t tt[6];
    uint4 pF;
    int nDstH = nDstHeight;
    int nDstW = nSrcWidth / 8;

    if (tid < nDstW && tidd < nDstH) {
        int j = tidd * nSrcWidth;
        int k = tid * 8;
        pF.x = (uint32_t)pV210[j + k + 0] + ((uint32_t)pV210[j + k + 1] << 16);
        pF.y = (uint32_t)pV210[j + k + 2] + ((uint32_t)pV210[j + k + 3] << 16);
        pF.z = (uint32_t)pV210[j + k + 4] + ((uint32_t)pV210[j + k + 5] << 16);
        pF.w = (uint32_t)pV210[j + k + 6] + ((uint32_t)pV210[j + k + 7] << 16);

        v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
        y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10) * 1164;
        u0 = (uint32_t)(pF.x & 0x000003FF);
        y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20) * 1164;
        u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
        y1 = (uint32_t)(pF.y & 0x000003FF) * 1164;
        u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
        y3 = (uint32_t)((pF.z & 0x000FFC00) >> 10) * 1164;
        v1 = (uint32_t)(pF.z & 0x000003FF);
        y5 = (uint32_t)((pF.w & 0x3FF00000) >> 20) * 1164;
        v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
        y4 = (uint32_t)(pF.w & 0x000003FF) * 1164;

        k = tid * 18;
        j *= 9;
        j /= 4;
        int r = 1596 * v0 - 891648, g = 813 * v0 + 392 * u0 - 542464, b = 2017 * u0 - 1107200;
        tt[0] = (y0 + r) / 1000;
        tt[1] = (y0 - g) / 1000;
        tt[2] = (y0 + b) / 1000;

        tt[3] = (y1 + r) / 1000;
        tt[4] = (y1 - g) / 1000;
        tt[5] = (y1 + b) / 1000;

        tt1[j + k + 0] = lookupTable[tt[0]];
        tt1[j + k + 1] = lookupTable[tt[1]];
        tt1[j + k + 2] = lookupTable[tt[2]];

        tt1[j + k + 3] = lookupTable[tt[3]];
        tt1[j + k + 4] = lookupTable[tt[4]];
        tt1[j + k + 5] = lookupTable[tt[5]];

        r = 1596 * v1 - 891648, g = 813 * v1 + 392 * u1 - 542464, b = 2017 * u1 - 1107200;
        tt[0] = (y2 + r) / 1000;
        tt[1] = (y2 - g) / 1000;
        tt[2] = (y2 + b) / 1000;

        tt[3] = (y3 + r) / 1000;
        tt[4] = (y3 - g) / 1000;
        tt[5] = (y3 + b) / 1000;

        tt1[j + k + 6] = lookupTable[tt[0]];
        tt1[j + k + 7] = lookupTable[tt[1]];
        tt1[j + k + 8] = lookupTable[tt[2]];

        tt1[j + k + 9] = lookupTable[tt[3]];
        tt1[j + k + 10] = lookupTable[tt[4]];
        tt1[j + k + 11] = lookupTable[tt[5]];

        r = 1596 * v2 - 891648, g = 813 * v2 + 392 * u2 - 542464, b = 2017 * u2 - 1107200;
        tt[0] = (y4 + r) / 1000;
        tt[1] = (y4 - g) / 1000;
        tt[2] = (y4 + b) / 1000;

        tt[3] = (y5 + r) / 1000;
        tt[4] = (y5 - g) / 1000;
        tt[5] = (y5 + b) / 1000;

        tt1[j + k + 12] = lookupTable[tt[0]];
        tt1[j + k + 13] = lookupTable[tt[1]];
        tt1[j + k + 14] = lookupTable[tt[2]];

        tt1[j + k + 15] = lookupTable[tt[3]];
        tt1[j + k + 16] = lookupTable[tt[4]];
        tt1[j + k + 17] = lookupTable[tt[5]];
    }
}

__global__ static void convertPToRGBKernel(const uint16_t *dpSrc, uint8_t *tt1, int nSrcWidth,
    int nDstWidth, int nDstHeight, int *lookupTable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t v0, y0, u0, y1;
    uint16_t tt[6];
    int nDstH = nDstHeight;
    int nDstW = nSrcWidth / 2;
    if (tid < nDstW && tidd < nDstH) {
        int k = tid * 2;
        int j = tidd * nSrcWidth;
        y0 = (uint32_t)dpSrc[j + k + 0] * 1164;
        y1 = (uint32_t)dpSrc[j + k + 1] * 1164;
        k = tid;
        j = tidd * nSrcWidth / 2 + nDstHeight * nSrcWidth;
        u0 = (uint32_t)dpSrc[j + k + 0];
        j = tidd * nSrcWidth / 2 + nDstHeight * nSrcWidth * 3 / 2;
        v0 = (uint32_t)dpSrc[j + k + 0];

        k = tid * 6;
        j = tidd * nDstWidth * 3;
        int r = 1596 * v0 - 891648, g = 813 * v0 + 392 * u0 - 542464, b = 2017 * u0 - 1107200;
        tt[0] = (y0 + r) / 1000;
        tt[1] = (y0 - g) / 1000;
        tt[2] = (y0 + b) / 1000;
        tt[3] = (y1 + r) / 1000;
        tt[4] = (y1 - g) / 1000;
        tt[5] = (y1 + b) / 1000;

        tt1[j + k + 0] = lookupTable[tt[0]];
        tt1[j + k + 1] = lookupTable[tt[1]];
        tt1[j + k + 2] = lookupTable[tt[2]];
        tt1[j + k + 3] = lookupTable[tt[3]];
        tt1[j + k + 4] = lookupTable[tt[4]];
        tt1[j + k + 5] = lookupTable[tt[5]];
    }
}

void convertToRGB(uint16_t *dpSrc, uint8_t *dpDst, int nSrcWidth, int nDstWidth, int nDstHeight,
    int *lookupTable, yuv_format yuvFormat, hipStream_t stream) {
    if (yuvFormat == PACKED) {
    dim3 blocks(32, 16, 1);
    dim3 grids((nSrcWidth + blocks.x - 1) / blocks.x, (nDstHeight + blocks.y - 1) / blocks.y, 1);
        convertVToRGBKernel << <grids, blocks, 0, stream >> > (dpSrc, dpDst, nSrcWidth, nDstWidth, nDstHeight, lookupTable);
    }
    else if (yuvFormat == PLANAR) {
    dim3 blocks(32, 32, 1);
    dim3 grids((nSrcWidth + blocks.x - 1) / blocks.x, (((nDstHeight * 2) + blocks.y) - 1) / blocks.y, 1);
        convertPToRGBKernel << <grids, blocks, 0, stream >> > (dpSrc, dpDst, nSrcWidth, nDstWidth, nDstHeight, lookupTable);
    }
}

__global__ static void convertToNppiKernel(uint16_t *dSrc, uint8_t *dDst,
	int nSrcWidth, int nDstWidth, int nDstHeight, int *lookupTable) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int tidd = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
	uint16_t tt[6];
	uint4 pF;
	int nDstH = nDstHeight;
	int nDstW = nSrcWidth / 8;

	if (tid < nDstW && tidd < nDstH) {
		int j = tidd * nSrcWidth;
		int k = tid * 8;
		pF.x = (uint32_t)dSrc[j + k + 0] + ((uint32_t)dSrc[j + k + 1] << 16);
		pF.y = (uint32_t)dSrc[j + k + 2] + ((uint32_t)dSrc[j + k + 3] << 16);
		pF.z = (uint32_t)dSrc[j + k + 4] + ((uint32_t)dSrc[j + k + 5] << 16);
		pF.w = (uint32_t)dSrc[j + k + 6] + ((uint32_t)dSrc[j + k + 7] << 16);

		v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
		y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10) * 1164;
		u0 = (uint32_t)(pF.x & 0x000003FF);
		y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20) * 1164;
		u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
		y1 = (uint32_t)(pF.y & 0x000003FF) * 1164;
		u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
		y3 = (uint32_t)((pF.z & 0x000FFC00) >> 10) * 1164;
		v1 = (uint32_t)(pF.z & 0x000003FF);
		y5 = (uint32_t)((pF.w & 0x3FF00000) >> 20) * 1164;
		v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
		y4 = (uint32_t)(pF.w & 0x000003FF) * 1164;

		k = tid * 18;
		j *= 9;
		j /= 4;
		int r = 1596 * v0 - 891648, g = 813 * v0 + 392 * u0 - 542464, b = 2017 * u0 - 1107200;
		tt[0] = (y0 + r) / 1000;
		tt[1] = (y0 - g) / 1000;
		tt[2] = (y0 + b) / 1000;

		tt[3] = (y1 + r) / 1000;
		tt[4] = (y1 - g) / 1000;
		tt[5] = (y1 + b) / 1000;

		dDst[j + k + 0] = lookupTable[tt[0]];
		dDst[j + k + 1] = lookupTable[tt[1]];
		dDst[j + k + 2] = lookupTable[tt[2]];

		dDst[j + k + 3] = lookupTable[tt[3]];
		dDst[j + k + 4] = lookupTable[tt[4]];
		dDst[j + k + 5] = lookupTable[tt[5]];

		r = 1596 * v1 - 891648, g = 813 * v1 + 392 * u1 - 542464, b = 2017 * u1 - 1107200;
		tt[0] = (y2 + r) / 1000;
		tt[1] = (y2 - g) / 1000;
		tt[2] = (y2 + b) / 1000;

		tt[3] = (y3 + r) / 1000;
		tt[4] = (y3 - g) / 1000;
		tt[5] = (y3 + b) / 1000;

		dDst[j + k + 6] = lookupTable[tt[0]];
		dDst[j + k + 7] = lookupTable[tt[1]];
		dDst[j + k + 8] = lookupTable[tt[2]];

		dDst[j + k + 9] = lookupTable[tt[3]];
		dDst[j + k + 10] = lookupTable[tt[4]];
		dDst[j + k + 11] = lookupTable[tt[5]];

		r = 1596 * v2 - 891648, g = 813 * v2 + 392 * u2 - 542464, b = 2017 * u2 - 1107200;
		tt[0] = (y4 + r) / 1000;
		tt[1] = (y4 - g) / 1000;
		tt[2] = (y4 + b) / 1000;

		tt[3] = (y5 + r) / 1000;
		tt[4] = (y5 - g) / 1000;
		tt[5] = (y5 + b) / 1000;

		dDst[j + k + 12] = lookupTable[tt[0]];
		dDst[j + k + 13] = lookupTable[tt[1]];
		dDst[j + k + 14] = lookupTable[tt[2]];

		dDst[j + k + 15] = lookupTable[tt[3]];
		dDst[j + k + 16] = lookupTable[tt[4]];
		dDst[j + k + 17] = lookupTable[tt[5]];
	}
}

void convertToRGBNpp(uint16_t *dSrc, uint8_t *dDst, int nSrcW, int nDstW, int nDstH,
	int *lookupTable, hipStream_t stream) {
	dim3 blocks(32, 16, 1);
	dim3 grids((nSrcW + blocks.x - 1) / blocks.x, (nDstH + blocks.y - 1) / blocks.y, 1);
	convertToNppiKernel <<<grids, blocks, 0, stream >> > (dSrc, dDst, nSrcW, nDstW, nDstH, lookupTable);
}