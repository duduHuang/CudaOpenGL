#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "resize.h"

__global__ static void resizeBatchKernel(const uint8_t *p_Src, int nSrcPitch, int nSrcHeight,
    uint8_t *p_dst, int nDstWidth, int nDstHeight) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uchar3 rgb;
    int nDstW = nDstWidth;
    int nDstH = nDstHeight;
    int yScale = nSrcHeight / nDstHeight;
    int xScale = 3 * (nSrcPitch / nDstWidth);
    if (tid < nDstW && tidd < nDstH) {
        int j = tidd * yScale * nSrcPitch * 3;
        int k = tid * xScale;
        rgb.x = p_Src[j + k + 0];
        rgb.y = p_Src[j + k + 1];
        rgb.z = p_Src[j + k + 2];
        k = tid * 3;
        j = tidd * nDstWidth * 3;
        p_dst[j + k + 0] = rgb.x;
        p_dst[j + k + 1] = rgb.y;
        p_dst[j + k + 2] = rgb.z;
    }
}

void resizeBatch(uint8_t *dpSrc, int nSrcPitch, int nSrcHeight, uint8_t *dpDst, int nDstWidth, int nDstHeight,
    hipStream_t stram) {
    dim3 blocks(32, 32, 1);
    dim3 grids((nSrcPitch + blocks.x - 1) / blocks.x, (((nSrcHeight * 3) + blocks.y) - 1) / blocks.y, 1);
    resizeBatchKernel << <grids, blocks, 0, stram >> > (dpSrc, nSrcPitch, nSrcHeight, dpDst, nDstWidth, nDstHeight);
}

__global__ static void resizeBatchKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight,
    uint16_t *p_dst, int nDstWidth, int nDstHeight) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint4 pF;
    int scale = nSrcHeight / nDstHeight;
    if (scale == 4) {
        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 6;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 32;

            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
            pF.w = (uint32_t)p_Src[j + k + 6];

            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);
            y1 = (uint32_t)(pF.w & 0x000003FF);

            pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
            pF.z = (uint32_t)p_Src[j + k + 12];

            y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            v1 = (uint32_t)(pF.z & 0x000003FF);

            pF.x = (uint32_t)p_Src[j + k + 16] + ((uint32_t)p_Src[j + k + 17] << 16);
            pF.z = ((uint32_t)p_Src[j + k + 21] << 16);
            pF.w = (uint32_t)p_Src[j + k + 22] + ((uint32_t)p_Src[j + k + 23] << 16);

            y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            y4 = (uint32_t)(pF.w & 0x000003FF);

            pF.y = ((uint32_t)p_Src[j + k + 27] << 16);

            y5 = (uint32_t)((pF.y & 0x3FF00000) >> 20);

            k = tid * 6;
            j = tidd * nDstWidth;
            p_dst[j + k + 0] = y0;
            p_dst[j + k + 1] = y1;
            p_dst[j + k + 2] = y2;
            p_dst[j + k + 3] = y3;
            p_dst[j + k + 4] = y4;
            p_dst[j + k + 5] = y5;
            k = tid * 3;
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight;
            p_dst[j + k + 0] = u0;
            p_dst[j + k + 1] = u1;
            p_dst[j + k + 2] = u2;
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            p_dst[j + k + 0] = v0;
            p_dst[j + k + 1] = v1;
            p_dst[j + k + 2] = v2;
        }
    } else if (scale == 6) {
        uint32_t v0, y0, u0, y1;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 2;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 16;
            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);

            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);

            pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);

            y1 = (uint32_t)((pF.x & 0x000FFC00) >> 10);

            k = tid * 2;
            j = tidd * nDstWidth;
            p_dst[j + k + 0] = y0;
            p_dst[j + k + 1] = y1;
            k = tid;
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight;
            p_dst[j + k + 0] = u0;
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            p_dst[j + k + 0] = v0;
        }
    } else if (scale == 2) {
        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 6;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 16;
            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
            pF.y = ((uint32_t)p_Src[j + k + 3] << 16);
            pF.z = ((uint32_t)p_Src[j + k + 5] << 16);
            pF.w = (uint32_t)p_Src[j + k + 6] + ((uint32_t)p_Src[j + k + 7] << 16);

            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);
            y1 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u1 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            v1 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            y2 = (uint32_t)(pF.w & 0x000003FF);

            pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
            pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
            pF.z = (uint32_t)p_Src[j + k + 12];
            pF.w = (uint32_t)p_Src[j + k + 14];

            y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            y4 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u2 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            v2 = (uint32_t)(pF.z & 0x000003FF);
            y5 = (uint32_t)(pF.w & 0x000003FF);

            k = tid * 6;
            j = tidd * nDstWidth;
            p_dst[j + k + 0] = y0;
            p_dst[j + k + 1] = y1;
            p_dst[j + k + 2] = y2;
            p_dst[j + k + 3] = y3;
            p_dst[j + k + 4] = y4;
            p_dst[j + k + 5] = y5;
            k = tid * 3;
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight;
            p_dst[j + k + 0] = u0;
            p_dst[j + k + 1] = u1;
            p_dst[j + k + 2] = u2;
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            p_dst[j + k + 0] = v0;
            p_dst[j + k + 1] = v1;
            p_dst[j + k + 2] = v2;
        }
    }
}

void resizeBatch(uint16_t *dpSrc, int nSrcPitch, int nSrcHeight, uint16_t *dpDst, int nDstWidth, int nDstHeight,
    hipStream_t stram) {
    dim3 blocks(32, 16, 1);
    dim3 grids((nSrcPitch + blocks.x - 1) / blocks.x, (nSrcHeight + blocks.y - 1) / blocks.y, 1);
    resizeBatchKernel << <grids, blocks, 0, stram >> > (dpSrc, nSrcPitch, nSrcHeight, dpDst, nDstWidth, nDstHeight);
}

__global__ static void resizeBatchKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight,
    uint8_t *dpDst, int nDstWidth, int nDstHeight, int *lookupTable_cuda) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint4 pF;
    int scale = nSrcHeight / nDstHeight;
    if (scale == 4) {
        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 6;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 32;

            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
            pF.w = (uint32_t)p_Src[j + k + 6];

            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);
            y1 = (uint32_t)(pF.w & 0x000003FF);

            pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
            pF.z = (uint32_t)p_Src[j + k + 12];

            y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            v1 = (uint32_t)(pF.z & 0x000003FF);

            pF.x = (uint32_t)p_Src[j + k + 16] + ((uint32_t)p_Src[j + k + 17] << 16);
            pF.z = ((uint32_t)p_Src[j + k + 21] << 16);
            pF.w = (uint32_t)p_Src[j + k + 22] + ((uint32_t)p_Src[j + k + 23] << 16);

            y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            y4 = (uint32_t)(pF.w & 0x000003FF);

            pF.y = ((uint32_t)p_Src[j + k + 27] << 16);

            y5 = (uint32_t)((pF.y & 0x3FF00000) >> 20);

            k = tid * 6;
            j = tidd * nDstWidth;
            dpDst[j + k + 0] = lookupTable_cuda[y0];
            dpDst[j + k + 1] = lookupTable_cuda[y1];
            dpDst[j + k + 2] = lookupTable_cuda[y2];
            dpDst[j + k + 3] = lookupTable_cuda[y3];
            dpDst[j + k + 4] = lookupTable_cuda[y4];
            dpDst[j + k + 5] = lookupTable_cuda[y5];
            k = tid * 3;
            j = tidd * nDstWidth / 2;
            dpDst[j + k + 0] = lookupTable_cuda[u0];
            dpDst[j + k + 1] = lookupTable_cuda[u1];
            dpDst[j + k + 2] = lookupTable_cuda[u2];
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            dpDst[j + k + 0] = lookupTable_cuda[v0];
            dpDst[j + k + 1] = lookupTable_cuda[v1];
            dpDst[j + k + 2] = lookupTable_cuda[v2];
        }
    }
    else if (scale == 6) {
        uint32_t v0, y0, u0, y1;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 2;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 16;
            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);

            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);

            pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);

            y1 = (uint32_t)((pF.x & 0x000FFC00) >> 10);

            k = tid * 2;
            j = tidd * nDstWidth;
            dpDst[j + k + 0] = lookupTable_cuda[y0];
            dpDst[j + k + 1] = lookupTable_cuda[y1];
            k = tid;
            j = tidd * nDstWidth / 2;
            dpDst[j + k + 0] = lookupTable_cuda[u0];
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            dpDst[j + k + 1] = lookupTable_cuda[v0];
        }
    }
    else if (scale == 2) {
        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 6;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 16;
            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
            pF.y = ((uint32_t)p_Src[j + k + 3] << 16);
            pF.z = ((uint32_t)p_Src[j + k + 5] << 16);
            pF.w = (uint32_t)p_Src[j + k + 6] + ((uint32_t)p_Src[j + k + 7] << 16);

            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);
            y1 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u1 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            v1 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            y2 = (uint32_t)(pF.w & 0x000003FF);

            pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
            pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
            pF.z = (uint32_t)p_Src[j + k + 12];
            pF.w = (uint32_t)p_Src[j + k + 14];

            y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            y4 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u2 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            v2 = (uint32_t)(pF.z & 0x000003FF);
            y5 = (uint32_t)(pF.w & 0x000003FF);

            k = tid * 6;
            j = tidd * nDstWidth;
            dpDst[j + k + 0] = lookupTable_cuda[y0];
            dpDst[j + k + 1] = lookupTable_cuda[y1];
            dpDst[j + k + 2] = lookupTable_cuda[y2];
            dpDst[j + k + 3] = lookupTable_cuda[y3];
            dpDst[j + k + 4] = lookupTable_cuda[y4];
            dpDst[j + k + 5] = lookupTable_cuda[y5];
            k = tid * 3;
            j = tidd * nDstWidth / 2;
            dpDst[j + k + 0] = lookupTable_cuda[u0];
            dpDst[j + k + 1] = lookupTable_cuda[u1];
            dpDst[j + k + 2] = lookupTable_cuda[u2];
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            dpDst[j + k + 0] = lookupTable_cuda[v0];
            dpDst[j + k + 1] = lookupTable_cuda[v1];
            dpDst[j + k + 2] = lookupTable_cuda[v2];
        }
    }
}

void resizeBatch(uint16_t *dpSrc, int nSrcPitch, int nSrcHeight, uint8_t *dpDst,
    int nDstWidth, int nDstHeight, int *lookupTable_cuda, hipStream_t stram) {
    dim3 blocks(32, 16, 1);
    dim3 grids((nSrcPitch + blocks.x - 1) / blocks.x, (nSrcHeight + blocks.y - 1) / blocks.y, 1);
    resizeBatchKernel << <grids, blocks, 0, stram >> > (dpSrc, nSrcPitch, nSrcHeight,
        dpDst, nDstWidth, nDstHeight, lookupTable_cuda);
}

__global__ static void resizeBatchKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight,
    uint8_t *dpDst0, uint8_t *dpDst1, uint8_t *dpDst2, int nDstWidth, int nDstHeight, int *lookupTable_cuda) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint4 pF;
    int scale = nSrcHeight / nDstHeight;
    if (scale == 4) {
        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 6;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 32;

            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
            pF.w = (uint32_t)p_Src[j + k + 6];

            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);
            y1 = (uint32_t)(pF.w & 0x000003FF);

            pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
            pF.z = (uint32_t)p_Src[j + k + 12];

            y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            v1 = (uint32_t)(pF.z & 0x000003FF);

            pF.x = (uint32_t)p_Src[j + k + 16] + ((uint32_t)p_Src[j + k + 17] << 16);
            pF.z = ((uint32_t)p_Src[j + k + 21] << 16);
            pF.w = (uint32_t)p_Src[j + k + 22] + ((uint32_t)p_Src[j + k + 23] << 16);

            y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            y4 = (uint32_t)(pF.w & 0x000003FF);

            pF.y = ((uint32_t)p_Src[j + k + 27] << 16);

            y5 = (uint32_t)((pF.y & 0x3FF00000) >> 20);

            k = tid * 6;
            j = tidd * nDstWidth;
            dpDst0[j + k + 0] = lookupTable_cuda[y0];
            dpDst0[j + k + 1] = lookupTable_cuda[y1];
            dpDst0[j + k + 2] = lookupTable_cuda[y2];
            dpDst0[j + k + 3] = lookupTable_cuda[y3];
            dpDst0[j + k + 4] = lookupTable_cuda[y4];
            dpDst0[j + k + 5] = lookupTable_cuda[y5];
            k = tid * 3;
            j = tidd * nDstWidth / 2;
            dpDst1[j + k + 0] = lookupTable_cuda[u0];
            dpDst1[j + k + 1] = lookupTable_cuda[u1];
            dpDst1[j + k + 2] = lookupTable_cuda[u2];
            dpDst2[j + k + 0] = lookupTable_cuda[v0];
            dpDst2[j + k + 1] = lookupTable_cuda[v1];
            dpDst2[j + k + 2] = lookupTable_cuda[v2];
        }
    } else if (scale == 6) {
        uint32_t v0, y0, u0, y1;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 2;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 16;
            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);

            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);

            pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);

            y1 = (uint32_t)((pF.x & 0x000FFC00) >> 10);

            k = tid * 2;
            j = tidd * nDstWidth;
            dpDst0[j + k + 0] = lookupTable_cuda[y0];
            dpDst0[j + k + 1] = lookupTable_cuda[y1];
            k = tid;
            j = tidd * nDstWidth / 2;
            dpDst1[j + k + 0] = lookupTable_cuda[u0];
            dpDst2[j + k + 1] = lookupTable_cuda[v0];
        }
    } else if (scale == 2) {
        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 6;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 16;
            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
            pF.y = ((uint32_t)p_Src[j + k + 3] << 16);
            pF.z = ((uint32_t)p_Src[j + k + 5] << 16);
            pF.w = (uint32_t)p_Src[j + k + 6] + ((uint32_t)p_Src[j + k + 7] << 16);

            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);
            y1 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u1 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            v1 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            y2 = (uint32_t)(pF.w & 0x000003FF);

            pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
            pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
            pF.z = (uint32_t)p_Src[j + k + 12];
            pF.w = (uint32_t)p_Src[j + k + 14];

            y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            y4 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u2 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            v2 = (uint32_t)(pF.z & 0x000003FF);
            y5 = (uint32_t)(pF.w & 0x000003FF);

            k = tid * 6;
            j = tidd * nDstWidth;
            dpDst0[j + k + 0] = lookupTable_cuda[y0];
            dpDst0[j + k + 1] = lookupTable_cuda[y1];
            dpDst0[j + k + 2] = lookupTable_cuda[y2];
            dpDst0[j + k + 3] = lookupTable_cuda[y3];
            dpDst0[j + k + 4] = lookupTable_cuda[y4];
            dpDst0[j + k + 5] = lookupTable_cuda[y5];
            k = tid * 3;
            j = tidd * nDstWidth / 2;
            dpDst1[j + k + 0] = lookupTable_cuda[u0];
            dpDst1[j + k + 1] = lookupTable_cuda[u1];
            dpDst1[j + k + 2] = lookupTable_cuda[u2];
            dpDst2[j + k + 0] = lookupTable_cuda[v0];
            dpDst2[j + k + 1] = lookupTable_cuda[v1];
            dpDst2[j + k + 2] = lookupTable_cuda[v2];
        }
    }
}

void resizeBatch(uint16_t *dpSrc, int nSrcPitch, int nSrcHeight, uint8_t *dpDst0, uint8_t *dpDst1, uint8_t *dpDst2,
    int nDstWidth, int nDstHeight, int *lookupTable_cuda, hipStream_t stram) {
    dim3 blocks(32, 16, 1);
    dim3 grids((nSrcPitch + blocks.x - 1) / blocks.x, (nSrcHeight + blocks.y - 1) / blocks.y, 1);
    resizeBatchKernel << <grids, blocks, 0, stram >> > (dpSrc, nSrcPitch, nSrcHeight,
        dpDst0, dpDst1, dpDst2, nDstWidth, nDstHeight, lookupTable_cuda);
}